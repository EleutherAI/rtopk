#include "hip/hip_runtime.h"
#include "rtopk_cuda.cuh"

// Convenience macros for error checking
#define CHECK_CUDA(x) AT_ASSERTM(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define RTOPK_CALL(DTYPE, CAST_DTYPE, W) \
    rtopk_kernel<DTYPE, W><<<blocks, threads, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>( \
        (DTYPE*)data.data_ptr<CAST_DTYPE>(), \
        (DTYPE*)values.data_ptr<CAST_DTYPE>(), \
        indices.data_ptr<int>(), \
        N, \
        dim_origin, \
        k, \
        max_iter, \
        precision_converted \
    )

// Wrapper function that launches the CUDA kernel
// Expects a 2D tensor 'data' of shape [N, dim_origin] and returns a tuple (values, indices),
// where for each of the N rows, the top-k approximate values (and their original indices)
// are stored in an output tensor of shape [N, k].
std::tuple<at::Tensor, at::Tensor> rtopk_forward_cuda(
    at::Tensor data,
    int64_t k,
    int64_t max_iter,
    double precision)
{
    // Ensure input is a contiguous CUDA tensor.
    CHECK_INPUT(data);
    AT_ASSERTM(data.dim() == 2, "data must be a 2D tensor");

    int N = data.size(0);
    int dim_origin = data.size(1);

    // Allocate output tensors.
    auto values = torch::empty({N, k}, data.options());
    auto indices = torch::empty({N, k}, torch::TensorOptions().dtype(torch::kInt32).device(data.device()));

    // Choose kernel launch parameters.
    // Each block processes WARPS_PER_BLOCK rows, with each warp (32 threads) handling one row.
    const int WARPS_PER_BLOCK = dim_origin < 1024 ? 8 : (dim_origin < 2048 ? 4 : (dim_origin < 4096 ? 2 : 1));
    const int threads = WARPS_PER_BLOCK * 32;
    const int blocks = (N + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    size_t shared_mem_size = WARPS_PER_BLOCK * dim_origin * data.element_size();

    // Launch the kernel based on data type.
    #ifdef __CUDA_BF16_TYPES_EXIST__
    if (data.scalar_type() == torch::kBFloat16) {
        __hip_bfloat16 precision_converted = __float2bfloat16(precision);
        if (WARPS_PER_BLOCK == 8) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 8);
        else if (WARPS_PER_BLOCK == 4) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 4);
        else if (WARPS_PER_BLOCK == 2) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 2);
        else RTOPK_CALL(__hip_bfloat16, at::BFloat16, 1);
    }
    else
    #endif
    if (data.scalar_type() == torch::kFloat32) {
        float precision_converted = precision;
        if (WARPS_PER_BLOCK == 8) RTOPK_CALL(float, float, 8);
        else if (WARPS_PER_BLOCK == 4) RTOPK_CALL(float, float, 4);
        else if (WARPS_PER_BLOCK == 2) RTOPK_CALL(float, float, 2);
        else RTOPK_CALL(float, float, 1);
    }
    else if (data.scalar_type() == torch::kInt16) {
        short precision_converted = precision;
        if (WARPS_PER_BLOCK == 8) RTOPK_CALL(short, short, 8);
        else if (WARPS_PER_BLOCK == 4) RTOPK_CALL(short, short, 4);
        else if (WARPS_PER_BLOCK == 2) RTOPK_CALL(short, short, 2);
        else RTOPK_CALL(short, short, 1);
    }
    else
    {
        throw std::invalid_argument("Unsupported data type. Only float32, bfloat16 and int16 are supported.");
    }

    // Check for any kernel launch errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel failed: ") + hipGetErrorString(err));
    }

    return std::make_tuple(values, indices);
}