#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rtopk_kernel.cuh"  // Make sure this file is in your include path

// Convenience macros for error checking
#define CHECK_CUDA(x) AT_ASSERTM(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define RTOPK_CALL(DTYPE, CAST_DTYPE, W) \
    rtopk_kernel<DTYPE, W><<<blocks, threads, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>( \
        (DTYPE*)data.data_ptr<CAST_DTYPE>(), \
        (DTYPE*)values.data_ptr<CAST_DTYPE>(), \
        indices.data_ptr<int>(), \
        N, \
        dim_origin, \
        k, \
        max_iter, \
        precision_converted \
    )

// Wrapper function that launches the CUDA kernel
// Expects a 2D tensor 'data' of shape [N, dim_origin] and returns a tuple (values, indices),
// where for each of the N rows, the top-k approximate values (and their original indices)
// are stored in an output tensor of shape [N, k].
std::tuple<torch::Tensor, torch::Tensor> rtopk_forward_cuda(
    torch::Tensor data,
    int k,
    int max_iter,
    float precision) 
{
    // Ensure input is a contiguous CUDA tensor.
    CHECK_INPUT(data);
    AT_ASSERTM(data.dim() == 2, "data must be a 2D tensor");

    int N = data.size(0);
    int dim_origin = data.size(1);

    // Allocate output tensors.
    auto values = torch::empty({N, k}, data.options());
    auto indices = torch::empty({N, k}, torch::TensorOptions().dtype(torch::kInt32).device(data.device()));

    // Choose kernel launch parameters.
    // Each block processes WARPS_PER_BLOCK rows, with each warp (32 threads) handling one row.
    const int WARPS_PER_BLOCK = dim_origin < 1024 ? 8 : (dim_origin < 2048 ? 4 : (dim_origin < 4096 ? 2 : 1));
    const int threads = WARPS_PER_BLOCK * 32;
    const int blocks = (N + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    size_t shared_mem_size = WARPS_PER_BLOCK * dim_origin * sizeof(float);

    // Launch the kernel based on data type.
    if (data.scalar_type() == torch::kFloat32) {
        float precision_converted = precision;
        if (WARPS_PER_BLOCK == 8) RTOPK_CALL(float, float, 8);
        else if (WARPS_PER_BLOCK == 4) RTOPK_CALL(float, float, 4);
        else if (WARPS_PER_BLOCK == 2) RTOPK_CALL(float, float, 2);
        else RTOPK_CALL(float, float, 1);
    }
#ifdef __CUDA_BF16_TYPES_EXIST__
    else if (data.scalar_type() == torch::kBFloat16) {
        __hip_bfloat16 precision_converted = __float2bfloat16(precision);
        if (WARPS_PER_BLOCK == 8) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 8);
        else if (WARPS_PER_BLOCK == 4) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 4);
        else if (WARPS_PER_BLOCK == 2) RTOPK_CALL(__hip_bfloat16, at::BFloat16, 2);
        else RTOPK_CALL(__hip_bfloat16, at::BFloat16, 1);
    }
#endif
    else {
        throw std::invalid_argument("Unsupported data type. Only float32 and bfloat16 are supported.");
    }

    // Check for any kernel launch errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel failed: ") + hipGetErrorString(err));
    }

    return std::make_tuple(values, indices);
}

std::tuple<torch::Tensor, torch::Tensor> rtopk_forward(
    torch::Tensor data, int k, int max_iter = 10, float precision = 1e-5) 
{
    return rtopk_forward_cuda(data, k, max_iter, precision);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("rtopk_forward", &rtopk_forward, "Approximate TopK forward");
}